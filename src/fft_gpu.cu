#include "fft_gpu.hpp"
// #pragma warning(disable:4505)
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <sstream>

static const char* cudaErrToStr( hipError_t e )
{
    return hipGetErrorString( e );
}

static const char* cufftErrToStr( hipfftResult r )
{
    switch ( r ) 
    {
        case HIPFFT_SUCCESS: return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN: return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED: return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE: return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED: return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED: return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE: return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";
        default: return "CUFFT_UNKNOWN_ERROR";
    }
}

bool gpu_fft(const std::vector<std::complex<double>>& in,
             std::vector<std::complex<double>>& out,
             bool inverse,
             std::string& err)
{
    out.resize( in.size() );
    const int n = static_cast<int>( in.size() );

    // Host bufor w formacie cuFFT (interleaved)
    std::vector<hipfftDoubleComplex> h_in( n ), h_out( n );
    for ( int i = 0; i < n; ++i )
    {
        h_in[i].x = in[i].real();
        h_in[i].y = in[i].imag();
    }

    hipfftDoubleComplex *d_in = nullptr, *d_out = nullptr;
    hipError_t cerr;
    hipfftResult r;

    cerr = hipMalloc( &d_in,  sizeof( hipfftDoubleComplex ) * n );
    if ( cerr != hipSuccess ) 
    {
        err = std::string("hipMalloc d_in: ") + cudaErrToStr( cerr ); 
        return false;
    }
    cerr = hipMalloc( &d_out, sizeof( hipfftDoubleComplex ) * n );
    if ( cerr != hipSuccess )
    {
        err = std::string("hipMalloc d_out: ") + cudaErrToStr( cerr ); hipFree( d_in ); 
        return false;
    }

    cerr = hipMemcpy( d_in, h_in.data(), sizeof( hipfftDoubleComplex ) * n, hipMemcpyHostToDevice );
    if ( cerr != hipSuccess )
    {
        err = std::string("hipMemcpy H2D: ") + cudaErrToStr( cerr );
        hipFree( d_in ); hipFree( d_out ); 
        return false;
    }

    hipfftHandle plan;
    r = hipfftPlan1d( &plan, n, HIPFFT_Z2Z, 1 );
    if ( r != HIPFFT_SUCCESS )
    {
        err = std::string("hipfftPlan1d: ") + cufftErrToStr( r );
        hipFree( d_in ); hipFree( d_out );
        return false;
    }

    r = hipfftExecZ2Z( plan, d_in, d_out, inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD );
    if ( r != HIPFFT_SUCCESS )
    {
        err = std::string("hipfftExecZ2Z: ") + cufftErrToStr( r );
        hipfftDestroy( plan ); hipFree( d_in ); hipFree( d_out );
        return false;
    }

    cerr = hipMemcpy( h_out.data(), d_out, sizeof( hipfftDoubleComplex ) * n, hipMemcpyDeviceToHost );
    if ( cerr != hipSuccess )
    {
        err = std::string("hipMemcpy D2H: ") + cudaErrToStr( cerr );
        hipfftDestroy( plan ); hipFree( d_in ); hipFree( d_out );
        return false;
    }

    hipfftDestroy( plan );
    hipFree( d_in );
    hipFree( d_out );

    for ( int i = 0; i < n; ++i )
    {
        out[i] = { h_out[i].x, h_out[i].y };
    }
    return true;
}
